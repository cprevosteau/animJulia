#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//	Dessin des ensembles de Julia
//	gr�ce � CUDA
//	avec une pr�cision float
////////////////////////////////////////////////////////////////////////////


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>

//Utilisation de SFML pour g�rer l'int�raction avec le clavier et afficher les images r�sultats
#include <SFML\Graphics.hpp>
#include <SFML\Window.hpp>
#include <SFML\System.hpp>

// Includes CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////


////// Cr�ation d'une classe pour les nombres complexes/////
	//Classe surtout utilis�e avec la version CPU de ce programme
	//mais tr�s peu dans ce rendu final

class juComplex
{
public:
		float r;
		float i;

	 juComplex():r(0),i(0){};
	 juComplex(float a,float b): r(a),i(b){};
	 float module2()
	 {
		return r*r+i*i;
	 }
	 juComplex operator+(juComplex const& a)
		{
			return juComplex(r+a.r,i+a.i);
		}
	juComplex operator*(juComplex const& a)
		{
			return juComplex(r*a.r-i*a.i,i*a.r+r*a.i);
		}

};






//////////////////////GPU PROGRAM

	/////D�claration  des objets qui seront stock�es dans la m�moire constante du GPU

		__constant__ unsigned int dim[2];				// Dimensions de l'image
		__constant__ float param[5];					// Param�tres arguments de la g�n�ration des ensembles de Julia :
														// 0 : Partie r�elle de la constante complexe
														// 1 : Partie imaginaire de la constate complexe
														// 2 : Position x de l'image
														// 3 : Position y de l'image
														// 4 : Echelle de l'image

		__constant__ unsigned char couleurDessin[3];	// Couleur RGB associ� aux points de l'ensemble de Julia
		__constant__ unsigned char couleurFond[3];		// Couleur RGB associ� au fond de l'image
		
	////Valeurs de C possibles par d�faut (lorsque l'image est r�initiallis�e
		const juComplex C[6]={juComplex(-0.7927,0.1609 ),juComplex(0.32,0.043),juComplex(-1.1380,0.2403),juComplex(-0.0986,-0.65186),juComplex(-0.1225,0.7449),juComplex(-0.3380,-0.6230)};



	////Fonctions min et max pour la gestion des couleurs

unsigned char maxCou(int a,int b)
{
	if(a>=b)
		return (unsigned char)a;
	else
		return (unsigned char)b;
}

unsigned char minCou(int a,int  b)
{
	if(a<=b)
		return (unsigned char)a;
	else
		return (unsigned char)b ;
}


	////Fonctions de mise � jour des param�tres selon l'enfoncement des touches

void gpRaff(float* t_param,unsigned char* t_couDes,unsigned char* t_couFond,sf::Time tdif)
{
	int vitesseD(500);			// vitesse de d�placement de l'image pour x et y
	float vitesseZoom(1.07);	// vitesse de zoom ou d�zoom
	int vitesseC(10);			// vitesse de changement de la constante complexe
	int vitesseCou(30);			// vitesse de changement de couleur

	//Modification des positions x et y en fonction de l'�chelle et du temps �coul� depuis la derni�re boucle tdif

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::Left))
			{
				t_param[2]-=(tdif.asSeconds())*vitesseD*t_param[4];
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::Right))
			{
				t_param[2]+=(tdif.asSeconds())*vitesseD*t_param[4];
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::Up))
			{
				t_param[3]-=(tdif.asSeconds())*vitesseD*t_param[4];
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::Down))
			{
				t_param[3]+=(tdif.asSeconds())*vitesseD*t_param[4];
			}

	//Modification de l'�chelle

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::Numpad8)||sf::Keyboard::isKeyPressed(sf::Keyboard::LShift))
			{
				t_param[4]/=vitesseZoom;
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::Numpad2)||sf::Keyboard::isKeyPressed(sf::Keyboard::LControl))
			{
				t_param[4]*=vitesseZoom;
			}

	// Modification de la constante complexe en fonction de l'�chelle et du temps �coul� tdif

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::Q))
			{
				if(t_param[0]<2)
					t_param[0]+=(tdif.asSeconds())*t_param[4]*vitesseC;
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::D))
			{
				if(t_param[0]>-2)
					t_param[0]-=(tdif.asSeconds())*t_param[4]*vitesseC;
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::Z))
			{
				if(t_param[1]<2)
					t_param[1]+=(tdif.asSeconds())*t_param[4]*vitesseC;
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::S))
			{
				if(t_param[1]>-2)
					t_param[1]-=(tdif.asSeconds())*t_param[4]*vitesseC;
			}


	// Modification de la couleur de dessin

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::R))
			{
				if(t_couDes[0]<255)
					t_couDes[0]=minCou(255,ceil(t_couDes[0]+tdif.asSeconds()*vitesseCou));
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::F))
			{
				if(t_couDes[0]>0)
					t_couDes[0]=maxCou(0,trunc(t_couDes[0]-tdif.asSeconds()*vitesseCou));
			
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::T))
			{
				if(t_couDes[1]<255)
					t_couDes[1]=minCou(255,ceil(t_couDes[1]+tdif.asSeconds()*vitesseCou));
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::G))
			{
				if(t_couDes[1]>0)
					t_couDes[1]=maxCou(0,trunc(t_couDes[1]-tdif.asSeconds()*vitesseCou));
			}
			if (sf::Keyboard::isKeyPressed(sf::Keyboard::Y))
			{
				if(t_couDes[2]<255)
					t_couDes[2]=minCou(255,ceil(t_couDes[2]+tdif.asSeconds()*vitesseCou));
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::H))
			{
				if(t_couDes[2]>0)
					t_couDes[2]=maxCou(0,trunc(t_couDes[2]-tdif.asSeconds()*vitesseCou));
			}


	// Modification de la couleur de fond
	
			if (sf::Keyboard::isKeyPressed(sf::Keyboard::U))
			{
				if(t_couFond[0]<255)
					t_couFond[0]=minCou(255,ceil(t_couFond[0]+tdif.asSeconds()*vitesseCou));
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::J))
			{
				if(t_couFond[0]>0)
					t_couFond[0]=maxCou(0,trunc(t_couFond[0]-tdif.asSeconds()*vitesseCou));
			
			}

				if (sf::Keyboard::isKeyPressed(sf::Keyboard::I))
			{
				if(t_couFond[1]<255)
					t_couFond[1]=minCou(255,ceil(t_couFond[1]+tdif.asSeconds()*vitesseCou));
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::K))
			{
				if(t_couFond[1]>0)
					t_couFond[1]=maxCou(0,trunc(t_couFond[1]-tdif.asSeconds()*vitesseCou));
			}
			if (sf::Keyboard::isKeyPressed(sf::Keyboard::O))
			{
				if(t_couFond[2]<255)
					t_couFond[2]=minCou(255,ceil(t_couFond[2]+tdif.asSeconds()*vitesseCou));
			}

			if (sf::Keyboard::isKeyPressed(sf::Keyboard::L))
			{
				if(t_couFond[2]>0)
					t_couFond[2]=maxCou(0,trunc(t_couFond[2]-tdif.asSeconds()*vitesseCou));
			}

}


//////// Fonction ex�cut�e sur le GPU qui renvoie l'intensit� de la couleur
	  // en fonction des coordonn�es (x,y) du complexe consid�r�

__device__ float gpJuCouleur(float const& x, float const& y)
{
	float zr(x);
	float zi(y);
	float zr0;
	int intens(0);

		while((zr*zr+zi*zi)<4 && intens<255) // Tant que le module carr� de Z est inf�rieur � 4
		{									 // avec une intensit� limit�e � 255
			zr0=zr;
			zr=zr*zr-zi*zi+param[0];		// Z re�oit Z^2 + C
			zi=2*zr0*zi+param[1];
			intens++;						// On ajoute de l'intensit�

		}
	return (float)(intens)/255.;
}


//////// Fonction kernel qui prend en argument le tableau de pixels stock�s sur le GPU
	  // et qui le remplit avec les valeurs associ�es aux param�tres et 
	  // la fonction gpJuCouleur
	  // Chaque thread correspondra au traitement d'un pixel.

__global__ void dessinKernel(sf::Uint8* dev_pTabPix)
{
	//On d�termine les coordonn�es du pixel
	// en fonction des coordonn�es du thread et du block
	// et des dimensions des blocks et de la grille

	const unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	const unsigned int j = blockIdx.y*blockDim.y+threadIdx.y;

	if(i<dim[0]&&j<dim[1])											// On v�rifie qu'on traite un thread correspondant � un pixel.
	{
			float x(param[2]+(i-(float)(dim[0])/2.)*(param[4]));  // On associe les coordonn�es d'un complexe au pixel
			float y(param[3]+(j-(float)(dim[1])/2.)*(param[4]));  // selon les param�tres.

			float pui(gpJuCouleur(x,y));							// On r�cup�re l'intensit� associ� � ce complexe.

			dev_pTabPix[(i+j*dim[0])*4]=round(pui*couleurDessin[0]+(1-pui)*couleurFond[0]);   // On associe la couleur au pixel
			dev_pTabPix[(i+j*dim[0])*4+1]=round(pui*couleurDessin[1]+(1-pui)*couleurFond[1]); // selon la couleurs de dessin
			dev_pTabPix[(i+j*dim[0])*4+2]=round(pui*couleurDessin[2]+(1-pui)*couleurFond[2]); // et celle de fond.
			dev_pTabPix[(i+j*dim[0])*4+3]=255;
	}

}

////////// Fonction qui prend en argument :
		// Un tableau de pixel sur l'h�te (ou "sur le CPU" en opposition � GPU)
		// Le tableau de pixels associ� allou� sur le GPU
		// Un tableau de param�tre sur l'h�te
		// Une couleur de dessin
		// Une couleur de fond
		// Une dimension d'image
		// La fonction remplit le tableau de pixels h�te avec ses nouvelles valeurs.

void gpDessiner(sf::Uint8* pTabPix,sf::Uint8* dev_pixels,float *cpu_param,unsigned char *cpu_couDes,unsigned char *cpu_couFond, unsigned int* cpu_dim)
{
	
	//// Mise � jour des variables constantes (sur le GPU) � partir de leurs homologues h�tes
		
		hipMemcpyToSymbol(HIP_SYMBOL(dim), cpu_dim, 2*sizeof(unsigned int));
		hipMemcpyToSymbol(HIP_SYMBOL(param), cpu_param, 5*sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(couleurDessin), cpu_couDes, 3*sizeof(unsigned char));
		hipMemcpyToSymbol(HIP_SYMBOL(couleurFond), cpu_couFond, 3*sizeof(unsigned char));

	//// D�finition des dimensions de la grille et des blocks de thread
		
		dim3 grille((cpu_dim[0]+15)/16,(cpu_dim[1]+15)/16); // Assure qu'il y ait au moins assez de thread pour traiter
		dim3 block(16,16);									// tous les pixels.


	// Appel � la fonction noyau qui va remplir le tableau de pixels GPU

		dessinKernel<<<grille,block>>>(dev_pixels);			


	// Mise � jour du tableau de pixels h�tes en fonction de celui GPU

		hipMemcpy( pTabPix, dev_pixels, cpu_dim[0]*cpu_dim[1] * 4*sizeof(sf::Uint8) , hipMemcpyDeviceToHost); 
}



int main()
{
	


	// Initiallisation

		bool active(true);		// Est-ce que la fen�tre est active ?
		bool ecriture(true);	// Ecriture dans la fen�tre (framerate et constante complexe utilis�e) ?

		sf::VideoMode video;			// R�cup�re la r�solution du bureau
		video=video.getDesktopMode();

		// On demande l'affichage voulu � l'utilisateur

		char choix;

		std::cout<<"Quelle r�solution voulez-vous ?"<<std::endl;
		std::cout<< "Rentrer R pour ensuite indiquer la r�solution, sinon rentrer n'importe quoi d'autre."<<std::endl;
		std::cin>>choix;

		bool pleinEcran(false);

		if(choix==char('R')||choix==char('r'))
			{
				int largeur;
				std::cout<<"Indiquer la largeur."<<std::endl;
				std::cin>>largeur;
				int hauteur;
				std::cout<<"Indiquer la hauteur."<<std::endl;
				std::cin>>hauteur;
				video.width=min(largeur,video.width);
				video.height=min(hauteur,video.height);
			}
			else
				pleinEcran=true;




		unsigned int* cpu_dim=new unsigned int[2]; //Dimensions de la fen�tre (ou de l'image)
		cpu_dim[0]=video.width;					
		cpu_dim[1]=video.height;

		sf::Texture texture;						//Texture de l'image
		texture.create(cpu_dim[0],cpu_dim[1]);		// avec les bonnes dimensions
		
		float* t_param =new float[5];				// Initiallisation du vecteur de param�tres
		t_param[0]=-0.7927;
		t_param[1]=0.1609;
		t_param[2]=0;
		t_param[3]=0;
		t_param[4]=0.01;

		unsigned char* t_couDes=new unsigned char[3]; // Initiallisation de la couleur de dessin
		t_couDes[0]=255;
		t_couDes[1]=255;
		t_couDes[2]=255;

		unsigned char* t_couFond=new unsigned char[3]; // Initiallisation de la couleur de fond
		t_couFond[0]=0;
		t_couFond[1]=0;
		t_couFond[2]=0;

		sf::Uint8* pixels = new sf::Uint8[cpu_dim[0]*cpu_dim[1] * 4];					// Allocation d'un tableau de pixels
		sf::Uint8* dev_pixels;															// Allocation de son homologue GPU
		hipMalloc( (void**)&dev_pixels, cpu_dim[0]*cpu_dim[1] * 4*sizeof(sf::Uint8) );

		gpDessiner(pixels,dev_pixels,t_param,t_couDes,t_couFond,cpu_dim );				// Mise � jour du tableau de pixels

	
		texture.update(pixels);								// Mise � jour de la texture 												
		sf::Sprite sprite;									// Cr�ation de l'objet associ� qui sera affich� dans la fen�tre
		sprite.setTexture(texture);
		sf::RenderWindow window;
		if(pleinEcran)
			window.create(video, "Ensemble de Julia",sf::Style::Fullscreen); //Cr�ation de la fen�tre
		else
			window.create(video, "Ensemble de Julia"); 

		window.setVerticalSyncEnabled(true);
		window.draw(sprite);											   //Desin de l'objet dans la fen�tre
		window.display();												   //Mise � jour de l'affichage de la fen�tre

	


		sf::Clock horloge;						//Initiallisation de l'horloge
		sf::Time t(horloge.getElapsedTime());	//Initiallisation des variables temps
		sf::Time t1;
		sf::Time tdif;

		sf::Text text;							//Initiallisation du texte � afficher dans la fen�tre
		sf::Font font;
		font.loadFromFile("arial.ttf");
		text.setFont(font);
		text.setCharacterSize(24);
		text.setColor(sf::Color::Color(255-t_couFond[0],255-t_couFond[1],255-t_couFond[2])); // Texte qui aura une couleur oppos�e � celle du fond

	
// On fait tourner le programme jusqu'� ce que la fen�tre soit ferm�e

		while (window.isOpen())
		{
		
			// On inspecte tous les �v�nements de la fen�tre qui ont �t� �mis depuis la pr�c�dente it�ration
			
			sf::Event event;
			while (window.pollEvent(event) || !active)
			{
			
			/* Tentative d'adapter la r�solution en fonction de l'agrandissement/r�duction de la fen�tre

				if (event.type == sf::Event::Resized)
				{
					
					cpu_dim[0]=event.size.width;
					cpu_dim[1]=event.size.height;

					delete pixels;
					hipFree( dev_pixels);
					pixels = new sf::Uint8[cpu_dim[0]*cpu_dim[1] * 4];
					hipMalloc( (void**)&dev_pixels, cpu_dim[0]*cpu_dim[1] * 4*sizeof(sf::Uint8) );
					texture.create(cpu_dim[0],cpu_dim[1]);
					window.create(sf::VideoMode(cpu_dim[0],cpu_dim[1]),"Ensemble de Julia");

				}
			*/

			// On d�termine si la fen�tre est active

				if (event.type == sf::Event::LostFocus)
						active=false;

				if (event.type == sf::Event::GainedFocus)
						active=true;


				// Ev�nement "fermeture demand�e" : on ferme la fen�tre

				if (event.type == sf::Event::Closed)
					window.close();

				if (event.type ==sf::Event::KeyPressed && active) // Si une touche est press�e et que la fen�tre est active
					{	
						switch(event.key.code)
						{
						case sf::Keyboard::A:					//Switch entre ecriture ou non du texte
							ecriture=!ecriture;
							break;
							
						case sf::Keyboard::Escape:				// On ferme la fen�tre
							window.close();
							break;

						case sf::Keyboard::Space:				//On r�initiallise la constante complexe et les couleurs
						{
								juComplex nC(C[rand()%6]);
								t_param[0]=nC.r;
								t_param[1]=nC.i;

								t_couFond[0]=(rand())%256;
								t_couFond[1]=(rand())%256;
								t_couFond[2]=(rand())%256;

								t_couDes[0]=(rand())%256;
								t_couDes[1]=(rand())%256;
								t_couDes[2]=(rand())%256;
						
								break;
						}
						default:
							break;
						}
					}
			}

			if(active)
			{
				//On r�cup�re le temps �coul� depuis la derni�re it�ration
					t1 = horloge.getElapsedTime();
					tdif=t1-t;
					t=t1;

				//On met � jour les param�tres
					gpRaff(t_param,t_couDes,t_couFond,tdif);

				//On met � jour les pixels et la fen�tres
					gpDessiner(pixels,dev_pixels,t_param,t_couDes,t_couFond,cpu_dim );
					texture.update(pixels);
					texture.setSmooth(true);
					window.clear();
					window.draw(sprite);

				// Si �criture, on affiche le framerate et la constante complexe utilis�e
					if(ecriture)
					{
						text.setColor(sf::Color::Color(255-t_couFond[0],255-t_couFond[1],255-t_couFond[2]));
						text.setString(std::to_string(1/tdif.asSeconds())+" "+std::to_string(t_param[0])+"+"+std::to_string(t_param[1])+"i");
						window.draw(text);
					}

				//Raffraichissement de la fen�tre
					window.display();
			}
		}

		//D�sallocation

		delete pixels;
		hipFree( dev_pixels);

	

    return 0;


}




